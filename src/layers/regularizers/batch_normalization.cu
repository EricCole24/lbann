#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// batch_normalization.cu - GPU helper routines for batch normalization layer
////////////////////////////////////////////////////////////////////////////////

#include "math.h"
#include <iostream>
#include "lbann/layers/regularizers/batch_normalization_cuda.hpp"
#include "lbann/utils/exception.hpp"

// Macros to check CUDA calls
#define FORCE_CHECK_CUDA(cuda_call)                                     \
  do {                                                                  \
    const hipError_t status = cuda_call;                               \
    if (status != hipSuccess) {                                        \
      std::cerr << "CUDA error: " << hipGetErrorString(status) << "\n"; \
      std::cerr << "Error at " << __FILE__ << ":" << __LINE__ << "\n";  \
      hipDeviceReset();                                                \
      throw lbann::lbann_exception("CUDA error");                       \
    }                                                                   \
  } while (0)
#ifdef LBANN_DEBUG
#define CHECK_CUDA(cuda_call) FORCE_CHECK_CUDA(cuda_call)
#else
#define CHECK_CUDA(cuda_call) cuda_call
#endif // #ifdef LBANN_DEBUG

// Atomic add functions
#if __CUDA_ARCH__ >= 530
__device__ inline __half atomic_add(__half* address, __half val) {
#if 0 // TODO: replace this once Nvidia implements atomicAdd for __half
  return atomicAdd(address, val);
#else
  unsigned int* address_as_uint = (unsigned int*) address;
  unsigned int old = *address_as_uint;
  __half* old_as_half = (__half*) &old;
  unsigned int assumed;
  unsigned int updated;
  __half* updated_as_half = (__half*) &updated;
  do {
    assumed = old;
    updated = old;
    *updated_as_half += value;
    old = atomicCAS(address_as_uint, assumed, updated);
  } while (assumed != old);
  return *old_as_half;
#endif // 0
}
#endif // __CUDA_ARCH__ >= 530
__device__ inline float atomic_add(float* address, float val) {
  return atomicAdd(address, val);
}
__device__ inline double atomic_add(double* address, double val) {
#if __CUDA_ARCH__ >= 600
  return atomicAdd(address, val);
#else
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
#endif // __CUDA_ARCH__ < 600
}

// Reciprocal square root functions
#if __CUDA_ARCH__ >= 530
__device__ inline float reciprocal_square_root(__half x) {
  return hrsqrt(x);
}
#endif // __CUDA_ARCH__ >= 530
__device__ inline float reciprocal_square_root(float x) {
  return rsqrtf(x);
}
__device__ inline double reciprocal_square_root(double x) {
  return rsqrt(x);
}

namespace lbann {
namespace batch_normalization_cuda {

template <typename DataType, int block_size>
__global__ void channel_sums_and_sqsums_kernel(
  int height,
  int width,
  int channel_size,
  const DataType * __restrict__ global_data,
        DataType * __restrict__ global_sums,
        DataType * __restrict__ global_sqsums) {

  // Indices
  const int tid = threadIdx.x;
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_sums[block_size];
  __shared__ DataType shared_sqsums[block_size];

  // Compute row sums in shared memory
  DataType sum = DataType(0);
  DataType sqsum = DataType(0);
  if(gidx < channel_size) {
    const int row = gidx + bidy * channel_size;
    for(int col = 0; col < width; ++col) {
      const DataType x = global_data[row + col * height];
      sum += x;
      sqsum += x * x;
    }
  }
  shared_sums[tid] = sum;
  shared_sqsums[tid] = sqsum;

  // Compute channel sum with shared memory reduction
  // TODO: unroll loops
  for(int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if(tid < stride) {
      shared_sums[tid] += shared_sums[tid + stride];
      shared_sqsums[tid] += shared_sqsums[tid + stride];
    }
  }

  // Output channel sum to global memory
  if(tid == 0) {
    atomic_add(&global_sums[bidy], shared_sums[0]);
    atomic_add(&global_sqsums[bidy], shared_sqsums[0]);
  }

}

template <typename DataType>
void channel_sums_and_sqsums(int height,
                             int width,
                             int num_channels,
                             const DataType *data_d,
                                   DataType *sums_d,
                                   DataType *sqsums_d,
                             hipStream_t stream) {
  
  // CUDA block size
  const int block_size = 256;

  // Clear GPU memory
  CHECK_CUDA(hipMemsetAsync(sums_d, 0, num_channels * sizeof(DataType), stream));
  CHECK_CUDA(hipMemsetAsync(sqsums_d, 0, num_channels * sizeof(DataType), stream));

  // Return if there is no input data
  if(width <= 0) return;

  // Launch CUDA kernel to compute sums and sums of squares
  const int channel_size = height / num_channels;
  dim3 block_dims, grid_dims;
  block_dims.x = block_size;
  grid_dims.x = (channel_size + block_size - 1) / block_size;
  grid_dims.y = num_channels;
  channel_sums_and_sqsums_kernel<DataType,block_size>
    <<<grid_dims, block_dims, 0, stream>>>
    (height, width, channel_size, data_d, sums_d, sqsums_d);

}

template <typename DataType>
__global__ void sums_to_statistics_kernel(
  int num_entries,
  DataType samples_per_sum,
  DataType decay,
  DataType * __restrict__ global_mean,
  DataType * __restrict__ global_var,
  DataType * __restrict__ global_running_mean,
  DataType * __restrict__ global_running_var) {
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  while(gid < num_entries) {

    // Compute statistics
    const DataType mean = global_mean[gid] / samples_per_sum;
    const DataType sqmean = global_var[gid] / samples_per_sum;
    DataType var = sqmean - mean * mean;
    var = var > DataType(0) ? var : DataType(0);
    var *= samples_per_sum / (samples_per_sum - DataType(1));
    global_mean[gid] = mean;
    global_var[gid] = var;

    // Compute running statistics
    DataType& running_mean = global_running_mean[gid];
    DataType& running_var = global_running_var[gid];
    running_mean = decay * running_mean + (DataType(1) - decay) * mean;
    running_var = decay * running_var + (DataType(1) - decay) * var;
    
    gid += blockDim.x * gridDim.x;
  }
}

template <typename DataType>
void sums_to_statistics(int num_entries,
                        int samples_per_sum,
                        DataType decay,
                        DataType *mean_d,
                        DataType *var_d,
                        DataType *running_mean_d,
                        DataType *running_var_d,
                        hipStream_t stream) {
  dim3 block_dims, grid_dims;
  block_dims.x = 256;
  grid_dims.x = (num_entries + block_dims.x - 1) / block_dims.x;
  sums_to_statistics_kernel<DataType>
    <<<grid_dims, block_dims, 0, stream>>>
    (num_entries, (DataType)samples_per_sum, decay,
     mean_d, var_d, running_mean_d, running_var_d);
}

template <typename DataType, int block_size>
__global__ void batch_normalization_kernel(
  int height,
  int width,
  int channel_size,
  const DataType * __restrict__ global_prev_activations,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_bias,
        DataType * __restrict__ global_activations) {

  // Indices
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const DataType mean = global_mean[bidy];
  const DataType var = global_var[bidy];
  const DataType scale = global_scale[bidy];
  const DataType bias = global_bias[bidy];

  // Get reciprocal of standard deviation
  const DataType inv_stdev = reciprocal_square_root(var + epsilon);

  // Apply batch normalization
  if(gidx < channel_size) {
    const int row = gidx + bidy * channel_size;
    for(int col = 0; col < width; ++col) {
      const DataType x = global_prev_activations[row + col * height];
      const DataType xhat = (x - mean) * inv_stdev;
      const DataType y = scale * xhat + bias;
      global_activations[row + col * height] = y;
    }
  }

}

template <typename DataType>
void batch_normalization(int height,
                         int width,
                         int num_channels,
                         const DataType *prev_activations_d,
                         const DataType *mean_d,
                         const DataType *var_d,
                         DataType epsilon,
                         const DataType *scale_d,
                         const DataType *bias_d,
                               DataType *activations_d,
                         hipStream_t stream) {

  // CUDA block size
  const int block_size = 256;

  // Return if there is no input data
  if(width <= 0) return;

  // Launch CUDA kernel to apply batch normalization
  const int channel_size = height / num_channels;
  dim3 block_dims, grid_dims;
  block_dims.x = block_size;
  grid_dims.x = (channel_size + block_size - 1) / block_size;
  grid_dims.y = num_channels;
  batch_normalization_kernel<DataType,block_size>
    <<<grid_dims, block_dims, 0, stream>>>
    (height, width, channel_size,
     prev_activations_d,
     mean_d, var_d, epsilon,
     scale_d, bias_d,
     activations_d);

}

template <typename DataType, int block_size>
__global__ void batch_normalization_backprop1_kernel(
  int height,
  int width,
  int channel_size,
  const DataType * __restrict__ global_prev_activations,
  const DataType * __restrict__ global_prev_error_signal,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
        DataType * __restrict__ global_dscale,
        DataType * __restrict__ global_dbias,
        DataType * __restrict__ global_dmean,
        DataType * __restrict__ global_dvar) {

  // Indices
  const int tid = threadIdx.x;
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_dscale[block_size];
  __shared__ DataType shared_dbias[block_size];
  __shared__ DataType shared_dmean[block_size];
  __shared__ DataType shared_dvar[block_size];

  // Copy batch normalization parameters to private memory
  const DataType mean = global_mean[bidy];
  const DataType var = global_var[bidy];
  const DataType scale = global_scale[bidy];

  // Compute useful constants
  const DataType inv_stdev = reciprocal_square_root(var + epsilon);
  const DataType dvar_factor = inv_stdev * inv_stdev * inv_stdev / 2;

  // Compute row-wise gradient contributions in shared memory
  DataType dscale = DataType(0);
  DataType dbias = DataType(0);
  DataType dmean = DataType(0);
  DataType dvar = DataType(0);
  if(gidx < channel_size) {
    const int row = gidx + bidy * channel_size;
    for(int col = 0; col < width; ++col) {
      const DataType x = global_prev_activations[row + col * height];
      const DataType xhat = (x - mean) * inv_stdev;
      const DataType dy = global_prev_error_signal[row + col * height];
      dscale += dy * xhat;
      dbias += dy;
      const DataType dxhat = dy * scale;
      dmean += - dxhat * inv_stdev;
      dvar += - dxhat * (x - mean) * dvar_factor;
    }
  }
  shared_dscale[tid] = dscale;
  shared_dbias[tid] = dbias;
  shared_dmean[tid] = dmean;
  shared_dvar[tid] = dvar;

  // Compute gradients with shared memory reduction
  // TODO: unroll loops
  for(int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if(tid < stride) {
      shared_dscale[tid] += shared_dscale[tid + stride];
      shared_dbias[tid] += shared_dbias[tid + stride];
      shared_dmean[tid] += shared_dmean[tid + stride];
      shared_dvar[tid] += shared_dvar[tid + stride];
    }
  }

  // Output channel sum to global memory
  if(tid == 0) {
    atomic_add(&global_dscale[bidy], shared_dscale[0]);
    atomic_add(&global_dbias[bidy], shared_dbias[0]);
    atomic_add(&global_dmean[bidy], shared_dmean[0]);
    atomic_add(&global_dvar[bidy], shared_dvar[0]);
  }

}

template <typename DataType>
void batch_normalization_backprop1(int height,
                                   int width,
                                   int num_channels,
                                   const DataType *prev_activations_d,
                                   const DataType *prev_error_signal_d,
                                   const DataType *mean_d,
                                   const DataType *var_d,
                                   DataType epsilon,
                                   const DataType *scale_d,
                                         DataType *dscale_d,
                                         DataType *dbias_d,
                                         DataType *dmean_d,
                                         DataType *dvar_d,
                                   hipStream_t stream) {
  
  // CUDA block size
  const int block_size = 256;

  // Clear GPU memory
  CHECK_CUDA(hipMemsetAsync(dscale_d, 0, num_channels * sizeof(DataType), stream));
  CHECK_CUDA(hipMemsetAsync(dbias_d, 0, num_channels * sizeof(DataType), stream));
  CHECK_CUDA(hipMemsetAsync(dmean_d, 0, num_channels * sizeof(DataType), stream));
  CHECK_CUDA(hipMemsetAsync(dvar_d, 0, num_channels * sizeof(DataType), stream));

  // Return if there is no input data
  if(width <= 0) return;

  // Launch CUDA kernel for first phase of batch normalization backward propagation
  const int channel_size = height / num_channels;
  dim3 block_dims, grid_dims;
  block_dims.x = block_size;
  grid_dims.x = (channel_size + block_size - 1) / block_size;
  grid_dims.y = num_channels;
  batch_normalization_backprop1_kernel<DataType,block_size>
    <<<grid_dims, block_dims, 0, stream>>>
    (height, width, channel_size,
     prev_activations_d, prev_error_signal_d,
     mean_d, var_d, epsilon, scale_d,
     dscale_d, dbias_d, dmean_d, dvar_d);

}

template <typename DataType, int block_size>
__global__ void batch_normalization_backprop2_kernel(
  int height,
  int local_width,
  int global_width,
  int channel_size,
  const DataType * __restrict__ global_prev_activations,
  const DataType * __restrict__ global_prev_error_signal,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_dmean,
  const DataType * __restrict__ global_dvar,
        DataType * __restrict__ global_error_signal) {

  // Indices
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const DataType mean = global_mean[bidy];
  const DataType var = global_var[bidy];
  const DataType scale = global_scale[bidy];
  const DataType dmean = global_dmean[bidy];
  const DataType dvar = global_dvar[bidy];

  // Compute useful constants
  const DataType inv_stdev = reciprocal_square_root(var + epsilon);
  const DataType dmean_term = dmean / (global_width * channel_size);
  const DataType dvar_term = dvar * 2 / (global_width * channel_size - 1);

  // Apply batch normalization
  if(gidx < channel_size) {
    const int row = gidx + bidy * channel_size;
    for(int col = 0; col < local_width; ++col) {
      const DataType x = global_prev_activations[row + col * height];
      const DataType dy = global_prev_error_signal[row + col * height];
      const DataType dxhat = dy * scale;
      DataType dx = dxhat * inv_stdev;
      dx += dmean_term;
      dx += dvar_term * (x - mean);
      global_error_signal[row + col * height] = dx;
    }
  }

}

template <typename DataType>
void batch_normalization_backprop2(int height,
                                   int local_width,
                                   int global_width,
                                   int num_channels,
                                   const DataType *prev_activations_d,
                                   const DataType *prev_error_signal_d,
                                   const DataType *mean_d,
                                   const DataType *var_d,
                                   DataType epsilon,
                                   const DataType *scale_d,
                                   const DataType *dmean_d,
                                   const DataType *dvar_d,
                                         DataType *error_signal_d,
                                   hipStream_t stream) {
  
  // CUDA block size
  const int block_size = 256;

  // Return if there is no input data
  if(local_width <= 0) return;

  // Launch CUDA kernel for second phase of batch normalization backward propagation
  const int channel_size = height / num_channels;
  dim3 block_dims, grid_dims;
  block_dims.x = block_size;
  grid_dims.x = (channel_size + block_size - 1) / block_size;
  grid_dims.y = num_channels;
  batch_normalization_backprop2_kernel<DataType,block_size>
    <<<grid_dims, block_dims, 0, stream>>>
    (height, local_width, global_width, channel_size,
     prev_activations_d, prev_error_signal_d,
     mean_d, var_d, epsilon, scale_d, dmean_d, dvar_d,
     error_signal_d);

}

// Explicit instantiation
template
void channel_sums_and_sqsums<float>(int height,
                                    int width,
                                    int num_channels,
                                    const float *data_d,
                                    float *sums_d,
                                    float *sqsums_d,
                                    hipStream_t stream);
template
void sums_to_statistics<float>(int num_entries,
                               int entries_per_sum,
                               float decay,
                               float *mean_d,
                               float *var_d,
                               float *running_mean_d,
                               float *running_var_d,
                               hipStream_t stream);
template
void batch_normalization<float>(int height,
                                int width,
                                int num_channels,
                                const float *prev_activations_d,
                                const float *mean_d,
                                const float *var_d,
                                float epsilon,
                                const float *scale_d,
                                const float *bias_d,
                                float *activations_d,
                                hipStream_t stream);
template
void batch_normalization_backprop1<float>(int height,
                                          int width,
                                          int num_channels,
                                          const float *prev_activations_d,
                                          const float *prev_error_signal_d,
                                          const float *mean_d,
                                          const float *var_d,
                                          float epsilon,
                                          const float *scale_d,
                                          float *dscale_d,
                                          float *dbias_d,
                                          float *dmean_d,
                                          float *dvar_d,
                                          hipStream_t stream);
template
void batch_normalization_backprop2<float>(int height,
                                          int local_width,
                                          int global_width,
                                          int num_channels,
                                          const float *prev_activations_d,
                                          const float *prev_error_signal_d,
                                          const float *mean_d,
                                          const float *var_d,
                                          float epsilon,
                                          const float *scale_d,
                                          const float *dmean_d,
                                          const float *dvar_d,
                                          float *error_signal_d,
                                          hipStream_t stream);
template
void channel_sums_and_sqsums<double>(int height,
                                     int width,
                                     int num_channels,
                                     const double *data_d,
                                     double *sums_d,
                                     double *sqsums_d,
                                     hipStream_t stream);
template
void sums_to_statistics<double>(int num_entries,
                                int entries_per_sum,
                                double decay,
                                double *mean_d,
                                double *var_d,
                                double *running_mean_d,
                                double *running_var_d,
                                hipStream_t stream);
template
void batch_normalization<double>(int height,
                                 int width,
                                 int num_channels,
                                 const double *prev_activations_d,
                                 const double *mean_d,
                                 const double *var_d,
                                 double epsilon,
                                 const double *scale_d,
                                 const double *bias_d,
                                 double *activations_d,
                                 hipStream_t stream);
template
void batch_normalization_backprop1<double>(int height,
                                           int width,
                                           int num_channels,
                                           const double *prev_activations_d,
                                           const double *prev_error_signal_d,
                                           const double *mean_d,
                                           const double *var_d,
                                           double epsilon,
                                           const double *scale_d,
                                           double *dscale_d,
                                           double *dbias_d,
                                           double *dmean_d,
                                           double *dvar_d,
                                           hipStream_t stream);
template
void batch_normalization_backprop2<double>(int height,
                                           int local_width,
                                           int global_width,
                                           int num_channels,
                                           const double *prev_activations_d,
                                           const double *prev_error_signal_d,
                                           const double *mean_d,
                                           const double *var_d,
                                           double epsilon,
                                           const double *scale_d,
                                           const double *dmean_d,
                                           const double *dvar_d,
                                           double *error_signal_d,
                                           hipStream_t stream);

} // namespace batch_normalization
} // namespace lbann

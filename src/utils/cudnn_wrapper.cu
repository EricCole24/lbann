#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// cudnn_wrapper .hpp .cpp - cuDNN support - wrapper classes, utility functions
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/cudnn_wrapper.hpp"
#include "lbann/utils/exception.hpp"

#include "El.hpp"

namespace lbann {
namespace cudnn {

namespace {

__global__ void constant_kernel(DataType *data,
                                DataType val,
                                El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  data[offset] = val;
}

__global__ void reduce_kernel(DataType *dst, const DataType *src,
                              El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  dst[offset] += src[offset];
}

#ifdef LBANN_HAS_NCCL2
__global__ void scale_kernel(DataType *data,
                             const DataType scale,
                             El::Int len) {

  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  data[offset] *= scale;

}
#endif // LBANN_HAS_NCCL2
}

void cudnn_manager::set_on_gpu(int i,
                               DataType* gpu_data,
                               DataType val,
                               int height,
                               int width) {
  CHECK_CUDA(hipSetDevice(m_gpus[i]));
  const El::Int len = height * width;
  const int tb_dim = 256;
  const int grid_dim = len/tb_dim + (len % tb_dim ? 1 : 0);
  constant_kernel<<<grid_dim, tb_dim>>>(gpu_data, val, len);
}

void cudnn_manager::allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                      El::Int height,
                                      El::Int width) {
  if (m_num_gpus < 2) {
    return;
  }

  // Determine work space size
  const El::Int work_space_size = get_minimum_work_space_size();
  const El::Int min_work_space_size = 1024;
  if (work_space_size < min_work_space_size) {
    std::stringstream err;
    err << __FILE__ << " " << __LINE__ << " :: "
        << "insufficient GPU work space "
        << "(requires " << min_work_space_size << " bytes on each GPU, "
        << "but only have " << work_space_size << " bytes)";
    throw lbann_exception(err.str());
  }

  // Setup work buffers
  const El::Int buf_len = work_space_size / (2 * sizeof(DataType));
  std::vector<DataType*> bufs[2];
  for(int i=0; i<m_num_gpus; ++i) {
    DataType* work_space = static_cast<DataType*>(get_work_space(i));
    bufs[0].push_back(work_space);
    bufs[1].push_back(work_space + buf_len);
  }  

  El::Int total_len = height * width;
  El::Int offset = 0;

  do {
    El::Int len = std::min(total_len - offset, buf_len);
    int sbuf_idx = 0;
    int dbuf_idx = 1;
    for (int j = 0; j < m_num_gpus - 1; ++j) {
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));
        int src_idx = i;
        int dst_idx = (i + 1) % m_num_gpus;
        int src_dev = m_gpus[src_idx];
        int dst_dev = m_gpus[dst_idx];
        DataType *src_buf = j == 0 ? gpu_data[src_idx] + offset : bufs[sbuf_idx][src_idx];
        DataType *dst_buf = bufs[dbuf_idx][dst_idx];
        // std::cerr << "Copying from device " << src_dev << " to device " << dst_dev << "\n";
        // copy to the next device in the ring
        FORCE_CHECK_CUDA(hipMemcpyPeerAsync(dst_buf, dst_dev, src_buf, src_dev,
                                             len * sizeof(DataType), get_stream(src_idx)));
      }
      synchronize();
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));
        DataType *dst_buf = bufs[dbuf_idx][i];
        // TODO: use Thrust
        int tb_dim = 256;
        int grid_dim = len / tb_dim + (len % tb_dim ? 1 : 0);
        reduce_kernel<<<grid_dim, tb_dim>>>(gpu_data[i] + offset, dst_buf, len);
      }
      std::swap(sbuf_idx, dbuf_idx);
    }
    offset += len;
  } while (offset < total_len);
}

/// @todo Efficient implementation
void cudnn_manager::global_allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                             El::Int height,
                                             El::Int width,
                                             El::mpi::Comm comm) {
  if(!is_nccl_used()){
    static Mat cpu_workspace;
    cpu_workspace.Resize(height, width, height);
    allreduce_on_gpus(gpu_data, height, width);
    copy_from_gpu(0, cpu_workspace, gpu_data[0]);
    synchronize();
    El::AllReduce(cpu_workspace, comm);
    broadcast_to_gpus(gpu_data, cpu_workspace);
  } else{
#ifdef LBANN_HAS_NCCL2
    global_allreduce_on_gpus_nccl (gpu_data, height, width);
    synchronize();
#else
    throw lbann_exception("cudnn_manager: NCCL not detected");
#endif // #ifdef LBANN_HAS_NCCL2
  }
}

#ifdef LBANN_HAS_NCCL2
/// Convert DataType to NCCL data type. DataType is either double or float (default).
ncclDataType_t cudnn_manager::nccl_datatype() {
  switch(sizeof(DataType) ) {
    case 8:
      return ncclDouble;
    case 4:
      return ncclFloat;
    case 2:
      return ncclHalf;
    default:
      throw lbann::lbann_exception("cudnn_wrapper_cuda: invalid data type for NCCL");
  }
}

void cudnn_manager::global_allreduce_on_gpus_nccl(std::vector<DataType*>& gpu_data,
                                                  El::Int height,
                                                  El::Int width,
                                                  DataType scale) {


/**
  gpu_data is a vector of pointers, each of which points to a part of
  matrix allocated to GPU memory. Since we assume that one MPI rank is
  assigned to one GPU, the number of element in gpu_data is 1. */

  int num_gpus_assigned = m_gpus.size();

  ncclDataType_t type = nccl_datatype();
  El::Int total_len = height * width;

  if(num_gpus_assigned > 1) ncclGroupStart();
  for(int i = 0; i < num_gpus_assigned; ++i) {
    CHECK_CUDA(hipSetDevice(m_gpus[i]));
    NCCLCHECK(ncclAllReduce(gpu_data[i], gpu_data[i], total_len, type, ncclSum, m_nccl_comm[i], get_stream(i)));

    /// Apply scaling, if scale != 1
    if(scale != DataType(1)) {
      int tb_dim = 256;
      int grid_dim = total_len/tb_dim + (total_len % tb_dim ? 1 : 0);
      scale_kernel<<<grid_dim, tb_dim>>>(gpu_data[i], scale, total_len);
    }
  }
  if(num_gpus_assigned > 1) ncclGroupEnd();
}
#endif // LBANN_HAS_NCCL2

} // namespace cudnn
} // namespace lbann

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// cudnn_wrapper .hpp .cpp - cuDNN support - wrapper classes, utility functions
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/cudnn_wrapper.hpp"
#include "lbann/utils/exception.hpp"

#include "El.hpp"

using namespace cudnn;
using namespace lbann;

namespace cudnn {

namespace {

__global__ void reduce_kernel(DataType *dst, const DataType *src,
                              El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  dst[offset] += src[offset];
}

__global__ void scale_kernel(DataType *data, 
                             const DataType scale,
                             El::Int len) {

  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  data[offset] *= scale;

}

}

void cudnn_manager::allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                      El::Int height,
                                      El::Int width) {
  if (m_num_gpus < 2) {
    return;
  }

  const El::Int buf_len = 1 << 27;
  const El::Int work_len = buf_len * 2; // double buffering
  const El::Int work_len_bytes = work_len * sizeof(DataType);

  std::vector<DataType*> bufs[2];
  for(int i=0; i<m_num_gpus; ++i) {
    if (get_work_space_size(i) < work_len_bytes) {
      set_work_space_size(i, work_len_bytes); 
    }
    bufs[0].push_back(static_cast<DataType*>(get_work_space(i)));
    bufs[1].push_back(static_cast<DataType*>(get_work_space(i)) + buf_len);
  }  

  El::Int total_len = height * width;
  El::Int offset = 0;

  do {
    El::Int len = std::min(total_len - offset, buf_len);
    int sbuf_idx = 0;
    int dbuf_idx = 1;
    for (int j = 0; j < m_num_gpus - 1; ++j) {
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));
        int src_dev = i;
        int dst_dev = (i + 1) % m_num_gpus;              
        DataType *src_buf = j == 0 ? gpu_data[src_dev] + offset : bufs[sbuf_idx][src_dev];
        DataType *dst_buf = bufs[dbuf_idx][dst_dev];
        // copy to the next device in the ring
        FORCE_CHECK_CUDA(hipMemcpyPeerAsync(dst_buf, dst_dev, src_buf, src_dev,
                                           len * sizeof(DataType), get_stream(src_dev)));
      }
      synchronize();
      for(int i = 0; i < m_num_gpus; ++i) {
        CHECK_CUDA(hipSetDevice(m_gpus[i]));        
        DataType *dst_buf = bufs[dbuf_idx][i];
        // TODO: use Thrust
        int tb_dim = 256;
        int grid_dim = len / tb_dim + (len % tb_dim ? 1 : 0);
        reduce_kernel<<<grid_dim, tb_dim>>>(gpu_data[i] + offset, dst_buf, len);
      }
      std::swap(sbuf_idx, dbuf_idx);
    }
    offset += len;
  } while (offset < total_len);
}

/// @todo Efficient implementation
void cudnn_manager::global_allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                             El::Int height,
                                             El::Int width,
                                             El::mpi::Comm comm) {
  if(!is_nccl_used()){
    static Mat cpu_workspace;
    cpu_workspace.Resize(height, width);
    allreduce_on_gpus(gpu_data, height, width);
    copy_from_gpu(0, cpu_workspace, gpu_data[0]);
    synchronize();
    El::AllReduce(cpu_workspace, comm);
    broadcast_to_gpus(gpu_data, cpu_workspace);
  } else{
#ifdef __LIB_NCCL
    global_allreduce_on_gpus_nccl (gpu_data, height, width);
    synchronize();
#else
    throw lbann_exception("cudnn_manager: NCCL not detected");
#endif // #ifdef __LIB_NCCL
  }
}

#ifdef __LIB_NCCL
/// Convert DataType to NCCL data type. DataType is either double or float (default).
ncclDataType_t cudnn_manager::nccl_datatype() {
  switch(sizeof(DataType) ) {
    case 8:
      return ncclDouble;
    case 4:
      return ncclFloat;
    case 2:
      return ncclHalf;
    default:
      throw lbann::lbann_exception("cudnn_wrapper_cuda: invalid data type for NCCL");
  }
}

void cudnn_manager::global_allreduce_on_gpus_nccl(std::vector<DataType*>& gpu_data,
                                                  El::Int height,
                                                  El::Int width,
                                                  DataType scale) {


/**
  gpu_data is a vector of pointers, each of which points to a part of
  matrix allocated to GPU memory. Since we assume that one MPI rank is
  assigned to one GPU, the number of element in gpu_data is 1. */

  int num_gpus_assigned = m_gpus.size();

  ncclDataType_t type = nccl_datatype();
  El::Int total_len = height * width;

  if(num_gpus_assigned > 1) ncclGroupStart();
  for(int i = 0; i < num_gpus_assigned; ++i) {
    CHECK_CUDA(hipSetDevice(m_gpus[i]));
    NCCLCHECK(ncclAllReduce(gpu_data[i], gpu_data[i], total_len, type, ncclSum, m_nccl_comm[i], get_stream(i)));

    /// Apply scaling, if scale != 1
    if(scale != DataType(1)) {
      int tb_dim = 256;
      int grid_dim = total_len/tb_dim + (total_len % tb_dim ? 1 : 0);
      scale_kernel<<<grid_dim, tb_dim>>>(gpu_data[i], scale, total_len);
    }
  }
  if(num_gpus_assigned > 1) ncclGroupEnd();
}
#endif // __LIB_NCCL

} // namespace cudnn
